#include "sh_handler.h"

#define DPDK_RING_SIZE (BATCH_SIZE * 8)
#define RTE_ETH_CRC_LEN 5
#define TOTAL_PKT_SIZE (PKT_SIZE + RTE_ETH_CRC_LEN)
#define ONELINE 6

#define GPU_TIME 1
#define MANI 0

unsigned char * pinned_pkt_buf;
unsigned char * tmp;
static int idx;

static uint64_t recv_total;

static uint64_t start;
static uint64_t end;

extern "C"
uint64_t monotonic_time() {
        struct timespec timespec;
        clock_gettime(CLOCK_MONOTONIC, &timespec);
        return timespec.tv_sec * ONE_SEC + timespec.tv_nsec;
}

extern "C"
void copy_to_gpu(unsigned char* buf, int nb_rx)
{
	hipMemcpy(pinned_pkt_buf + (idx * BATCH_SIZE), buf, sizeof(unsigned char)*BATCH_SIZE, hipMemcpyHostToDevice);

	idx++;
	if(idx == 512)
		idx = 0;

#if GPU_TIME
	end = monotonic_time();
	recv_total += nb_rx;

	if(end - start >= ONE_SEC)
	{	
		printf("recv_total = %ld\n", recv_total);
		recv_total = 0;
		start = monotonic_time();
	}
#endif
}

extern "C"
void set_gpu_mem_for_dpdk(void)
{
	size_t pkt_buffer_size = DPDK_RING_SIZE;
	idx = 0;
	ASSERTRT(hipMalloc((void**)&pinned_pkt_buf, pkt_buffer_size));
  	ASSERTRT(hipMemset(pinned_pkt_buf, 0, pkt_buffer_size));

	ASSERTRT(hipMalloc((void**)&tmp, sizeof(unsigned char) * 6));
  	ASSERTRT(hipMemset(tmp, 0, 6));

#if GPU_TIME
	start = monotonic_time();
	
	recv_total = 0;
#endif

	START_GRN
	printf("[Done]____GPU mem set for dpdk____\n");
	END
}

