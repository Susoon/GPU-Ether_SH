#include "hip/hip_runtime.h"
#include "sh_handler.h"

#define RING_BATCH_SIZE 8
#define DPDK_RING_SIZE (BATCH_SIZE * RING_BATCH_SIZE)
#define RTE_ETH_CRC_LEN 5
#define TOTAL_PKT_SIZE (PKT_SIZE + RTE_ETH_CRC_LEN)
#define ONELINE 6

#define GPU_TIME 1
#define MANI 0

unsigned char * pinned_pkt_buf;
unsigned char * tmp;
static int idx;

uint64_t start;
uint64_t end;

static uint64_t recv_total;

__global__ void print_gpu(unsigned char* d_pkt_buf)
{
	int i;
	START_RED
	printf("[GPU]:\n");
	for(i = 0; i < TOTAL_PKT_SIZE; i++)
	{
		if(i != 0 && i % ONELINE ==0)
			printf("\n");
		printf("%02x ", d_pkt_buf[i]);
	}
	printf("\n");
	END
}

__global__ void mani_pkt_gpu(unsigned char * d_pkt_buf, unsigned char * tmp, uint64_t * recv_total, int size)
{
	*recv_total += 1;
	printf("recv_total = %ld\n", *recv_total);
	int i;
	// Swap mac
	for(i = 0; i < 6; i++){
		tmp[i] = d_pkt_buf[i];
		d_pkt_buf[i] = d_pkt_buf[i + 6];
		d_pkt_buf[i + 6] = tmp[i];
	}
	// Swap ip
	for(i = 26; i < 30; i++){
		tmp[i-26] = d_pkt_buf[i];
		d_pkt_buf[i] = d_pkt_buf[i + 4];
		d_pkt_buf[i + 4] = tmp[i-26];
	}
	// Swap port
	for(i = 34; i < 36; i++){
		tmp[i-34] = d_pkt_buf[i];
		d_pkt_buf[i] = d_pkt_buf[i + 2];
		d_pkt_buf[i + 2] = tmp[i-34];
	}
	//Manipulatate data
	for(i = 36; i < size; i++){
		d_pkt_buf[i] = 0;
	}
}

extern "C"
uint64_t monotonic_time() {
        struct timespec timespec;
        clock_gettime(CLOCK_MONOTONIC, &timespec);
        return timespec.tv_sec * ONE_SEC + timespec.tv_nsec;
}

extern "C"
void copy_to_gpu(unsigned char* buf, int size, float * total_time)
{
	hipEvent_t go, stop;
	float time = 0;
	hipEventCreate(&go);
	hipEventCreate(&stop);

	hipMemcpy(pinned_pkt_buf + (idx * BATCH_SIZE), buf, sizeof(unsigned char)*size, hipMemcpyHostToDevice);

	idx++;
	if(idx == 512)
		idx = 0;

#if MANI
	hipEventRecord(go, 0);
	//for(int i = 0; i < BATCH_NUM; i += PKT_SIZE)
	//{
//		mani_pkt_gpu<<<1,1>>>(pinned_pkt_buf + (idx * BATCH_SIZE), tmp, &recv_total, size);
//	}
	//print_gpu<<<1,1>>>(pinned_pkt_buf + (idx * BATCH_SIZE));
	hipDeviceSynchronize();
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, go, stop);
	
	hipEventDestroy(go);
	hipEventDestroy(stop);

	*total_time += time;
	time = 0;
	
	if(*total_time >= 10)
	{
		//printf("recv_total = %ld\n", recv_total);
		//printf("total_time = %f\n", *total_time);
		*total_time = 0;
		recv_total = 0;
	}

	hipMemcpy(buf, pinned_pkt_buf + (idx * BATCH_SIZE), sizeof(unsigned char) * size, hipMemcpyDeviceToHost);
#endif

#if GPU_TIME
	for(int i = 0; i < BATCH_NUM; i += PKT_SIZE)
	{
		mani_pkt_gpu<<<1,1>>>(pinned_pkt_buf + (idx * BATCH_SIZE) + i, tmp, &recv_total, size);
	}
	print_gpu<<<1,1>>>(pinned_pkt_buf + (idx * BATCH_SIZE));
	hipDeviceSynchronize();
	end = monotonic_time();

	if(end - start >= ONE_SEC)
	{	
		printf("recv_total = %ld\n", recv_total);
		recv_total = 0;
		start = monotonic_time();
	}
#endif
}

extern "C"
void set_gpu_mem_for_dpdk(void)
{
	size_t pkt_buffer_size = DPDK_RING_SIZE;
	idx = 0;
	ASSERTRT(hipMalloc((void**)&pinned_pkt_buf, pkt_buffer_size));
  	ASSERTRT(hipMemset(pinned_pkt_buf, 1, pkt_buffer_size));

	ASSERTRT(hipMalloc((void**)&tmp, sizeof(unsigned char) * 6));
  	ASSERTRT(hipMemset(tmp, 0, 6));

#if GPU_TIME
	start = monotonic_time();
	
	recv_total = 0;
#endif

	START_GRN
	printf("[Done]____GPU mem set for dpdk____\n");
	END
}

__global__ void read_loop(void)
{
}

extern "C"
void read_handler(void)
{
	read_loop<<<1,1>>>();
	hipDeviceSynchronize();
}

