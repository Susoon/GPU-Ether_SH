#include "hip/hip_runtime.h"
#include "sh_handler.h"

#define RING_BATCH_SIZE 8
#define DPDK_RING_SIZE (BATCH_SIZE * RING_BATCH_SIZE)
#define RTE_ETH_CRC_LEN 5
#define TOTAL_PKT_SIZE (PKT_SIZE + RTE_ETH_CRC_LEN)
#define ONELINE 6

#define GPU_TIME 1
#define MANI 0

unsigned char * pinned_pkt_buf;
unsigned char * tmp;
static int idx;
static unsigned int * pkt_cnt;
static unsigned int cur_pkt;

__global__ void print_gpu(unsigned char* d_pkt_buf)
{
	int i;
	START_RED
	printf("[GPU]:\n");
	for(i = 0; i < TOTAL_PKT_SIZE; i++)
	{
		if(i != 0 && i % ONELINE ==0)
			printf("\n");
		printf("%02x ", d_pkt_buf[i]);
	}
	printf("\n");
	END
}

__global__ void mani_pkt_gpu(unsigned char * d_pkt_buf, unsigned char * tmp, unsigned int *pkt_cnt, int size)
{
	atomicAdd(pkt_cnt, size);
	int i;
	// Swap mac
	for(i = 0; i < 6; i++){
		tmp[i] = d_pkt_buf[i];
		d_pkt_buf[i] = d_pkt_buf[i + 6];
		d_pkt_buf[i + 6] = tmp[i];
	}
	// Swap ip
	for(i = 26; i < 30; i++){
		tmp[i-26] = d_pkt_buf[i];
		d_pkt_buf[i] = d_pkt_buf[i + 4];
		d_pkt_buf[i + 4] = tmp[i-26];
	}
	// Swap port
	for(i = 34; i < 36; i++){
		tmp[i-34] = d_pkt_buf[i];
		d_pkt_buf[i] = d_pkt_buf[i + 2];
		d_pkt_buf[i + 2] = tmp[i-34];
	}
	//Manipulatate data
	for(i = 36; i < size; i++){
		d_pkt_buf[i] = 0;
	}
}

extern "C"
uint64_t monotonic_time() {
        struct timespec timespec;
        clock_gettime(CLOCK_MONOTONIC, &timespec);
        return timespec.tv_sec * ONE_SEC + timespec.tv_nsec;
}

extern "C"
int copy_to_gpu(unsigned char* buf, int size, float * total_time)
{
	hipMemcpy(pinned_pkt_buf + (idx * BATCH_SIZE), buf, sizeof(unsigned char)*size, hipMemcpyHostToDevice);

	idx++;
	if(idx == 512)
		idx = 0;

#if GPU_TIME
	
	mani_pkt_gpu<<<1,1>>>(pinned_pkt_buf + (idx * BATCH_SIZE), tmp, pkt_cnt, size);
	//print_gpu<<<1,1>>>(pinned_pkt_buf + (idx * BATCH_SIZE));
	hipDeviceSynchronize();

	hipMemcpy(buf, pinned_pkt_buf + (idx * BATCH_SIZE), sizeof(unsigned char) * size, hipMemcpyDeviceToHost);
	hipMemcpy(&cur_pkt, pkt_cnt, sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemset(pkt_cnt, 0, sizeof(unsigned int));

	return cur_pkt;
#endif
}

extern "C"
void set_gpu_mem_for_dpdk(void)
{
	size_t pkt_buffer_size = DPDK_RING_SIZE;
	idx = 0;
	ASSERTRT(hipMalloc((void**)&pinned_pkt_buf, pkt_buffer_size));
  	ASSERTRT(hipMemset(pinned_pkt_buf, 1, pkt_buffer_size));

	ASSERTRT(hipMalloc((void**)&tmp, sizeof(unsigned char) * 6));
  	ASSERTRT(hipMemset(tmp, 0, 6));

	ASSERTRT(hipMalloc((void**)&pkt_cnt, sizeof(unsigned int)));
  	ASSERTRT(hipMemset(pkt_cnt, 0, sizeof(unsigned int)));

#if GPU_TIME
	//start = monotonic_time();
#endif

	START_GRN
	printf("[Done]____GPU mem set for dpdk____\n");
	END
}
