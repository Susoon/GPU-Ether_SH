#include "hip/hip_runtime.h"
#include "sh_handler.h"

#define RING_BATCH_SIZE 8
#define DPDK_RING_SIZE (BATCH_SIZE * RING_BATCH_SIZE)
#define RTE_ETH_CRC_LEN 5
#define TOTAL_PKT_SIZE (PKT_SIZE + RTE_ETH_CRC_LEN)
#define ONELINE 6
#define DUMP 0

unsigned char * pinned_pkt_buf;
unsigned char * tmp;
static int idx;
static unsigned int * pkt_cnt;
static unsigned int cur_pkt;

__global__ void print_gpu(unsigned char* d_pkt_buf)
{
	if(d_pkt_buf[PKT_SIZE - 1] != 0)
		return;
	int i;
	START_RED
	printf("[GPU]:\n");
	for(i = 0; i < BATCH_SIZE; i++)
	{
		if(i != 0 && i % ONELINE == 0)
			printf("\n");
		if(i != 0 && i % PKT_SIZE == 0)
			printf("\n");
		printf("%02x ", d_pkt_buf[i]);
	}
	printf("\n");
	END
}

__global__ void mani_pkt_gpu(unsigned char * d_pkt_buf, unsigned char * tmp, unsigned int *pkt_cnt, int size)
{
	atomicAdd(pkt_cnt, size);
	printf("MANI!!!\n");

	int i;
	// Swap mac
	for(i = 0; i < 6; i++){
		tmp[i] = d_pkt_buf[i];
		d_pkt_buf[i] = d_pkt_buf[i + 6];
		d_pkt_buf[i + 6] = tmp[i];
	}
	// Swap ip
	for(i = 26; i < 30; i++){
		tmp[i-26] = d_pkt_buf[i];
		d_pkt_buf[i] = d_pkt_buf[i + 4];
		d_pkt_buf[i + 4] = tmp[i-26];
	}
	// Swap port
	for(i = 34; i < 36; i++){
		tmp[i-34] = d_pkt_buf[i];
		d_pkt_buf[i] = d_pkt_buf[i + 2];
		d_pkt_buf[i + 2] = tmp[i-34];
	}	
	//Manipulatate data
	for(i = 36; i < size; i++){
		d_pkt_buf[i] = 0;
	}
}

extern "C"
uint64_t monotonic_time() {
        struct timespec timespec;
        clock_gettime(CLOCK_MONOTONIC, &timespec);
        return timespec.tv_sec * ONE_SEC + timespec.tv_nsec;
}

extern "C"
int copy_to_gpu(unsigned char* buf, int size)
{
	hipMemcpy(pinned_pkt_buf + (idx * BATCH_SIZE), buf, sizeof(unsigned char)* PKT_SIZE * size, hipMemcpyHostToDevice);

	idx++;
	if(idx == 512)
		idx = 0;

	mani_pkt_gpu<<<1,1>>>(pinned_pkt_buf + (idx * BATCH_SIZE), tmp, pkt_cnt, size);
#if DUMP
	print_gpu<<<1,1>>>(pinned_pkt_buf + (idx * BATCH_SIZE));
#endif
	hipDeviceSynchronize();

	hipMemcpy(buf, pinned_pkt_buf + (idx * BATCH_SIZE), sizeof(unsigned char) * size, hipMemcpyDeviceToHost);
	hipMemcpy(&cur_pkt, pkt_cnt, sizeof(unsigned int), hipMemcpyDeviceToHost);
	hipMemset(pkt_cnt, 0, sizeof(unsigned int));

	return cur_pkt;
}

extern "C"
void set_gpu_mem_for_dpdk(void)
{
	size_t pkt_buffer_size = DPDK_RING_SIZE;
	idx = 0;
	ASSERTRT(hipMalloc((void**)&pinned_pkt_buf, pkt_buffer_size));
  	ASSERTRT(hipMemset(pinned_pkt_buf, 1, pkt_buffer_size));

	ASSERTRT(hipMalloc((void**)&tmp, sizeof(unsigned char) * 6));
  	ASSERTRT(hipMemset(tmp, 0, 6));

	ASSERTRT(hipMalloc((void**)&pkt_cnt, sizeof(unsigned int)));
  	ASSERTRT(hipMemset(pkt_cnt, 0, sizeof(unsigned int)));

	START_GRN
	printf("[Done]____GPU mem set for dpdk____\n");
	END
}

__global__ void monitoring_loop(unsigned char * d_pkt_buf, int ){
	START_GRN
		printf("Control is returned to CPU! :)\n");
	END
	int prev_pkt[2] = {0,}, cur_pkt[2] = {0,};
	double pkts[2];
	char units[] = {' ', 'K', 'M', 'G', 'T'};
	char pps[2][40];
	char bps[2][40];
	int buf_idx[512] = {0,};
	int p_size=0;
	int i, j;

	uint64_t last_stats_printed = monotonic_time();
	uint64_t time;
	
	while(true)                                           
	{
		time = monotonic_time();
		if(time - last_stats_printed > 1000 * 1000 * 1000){

			last_stats_printed = time;

			hipError_t err = hipMemcpy(&cur_pkt[0], &pkt_cnt[0], sizeof(int), hipMemcpyDeviceToHost);
			hipError_t err2 = hipMemcpy(&cur_pkt[1], &pkt_cnt[1], sizeof(int), hipMemcpyDeviceToHost);
			hipError_t err3 = hipMemcpy(&p_size, pkt_size, sizeof(int), hipMemcpyDeviceToHost);
			hipError_t err4 = hipMemcpy(buf_idx, (p_buf->rx_buf_idx), sizeof(int)*512, hipMemcpyDeviceToHost);


			// CKJUNG, 18.08.07 For check
			//printf("Error-code of hipMemcpy: %d\n", err);
			//if(err != hipSuccess || err2 != hipSuccess || err3 != hipSuccess || err4 != hipSuccess || err5 != hipSuccess)
			if(err != hipSuccess || err2 != hipSuccess || err3 != hipSuccess || err4 != hipSuccess)
			{
				printf("hipMemcpy, pkt_cnt or buf_idx, error!\n");
			}
			system("clear");	
#if 0
			printf("[CKJUNG] buf #0\n");
			for(i = 0; i < 1024; i++){
				printf("%d ", data[i]);
			}
			printf("\n\n");
#endif
			for(i = 0; i < 2; i++){
				double tmp_pps;
				double tmp;
				//double batch;
				if (prev_pkt[i] != cur_pkt[i]){ // If we got a traffic flow
					pkts[i] = (double)(cur_pkt[i] - prev_pkt[i]);

#if 0
					if(i == 0)
						printf("RX_pkts: %d\n", (int)pkts[i]); 
					else
						printf("TX_pkts: %d\n", (int)pkts[i]); 
#endif
					tmp = tmp_pps = pkts[i];
					//batch = tmp/BATCH;
					for(j = 0; tmp >= 1000 && j < sizeof(units)/sizeof(char) -1; j++)
						tmp /= 1000;
					sprintf(pps[i],"%.3lf %c" ,tmp, units[j]);
#if TX
					p_size = PKT_SIZE;
#endif

					//tmp = pkts[i] * p_size * 8; // Bytes -> Bits
					tmp = pkts[i] * p_size * 8 + tmp_pps * 20 * 8; // Add IFG also, 20.01.15, CKJUNG
					for(j = 0; tmp >= 1000 && j < sizeof(units)/sizeof(char) -1; j++)
						tmp /= 1000;
					sprintf(bps[i],"%.3lf %c" ,tmp, units[j]);

					if(i == 0)
						printf("[RX] pps: %spps %sbps, pkt_size: %d \n", pps[i], bps[i], p_size);
					else{
						printf("[TX] pps: %spps %sbps, pkt_size: %d \n", pps[i], bps[i], p_size);
					}
				}else{
					if(i == 0)
						printf("[RX] pps: None\n");
					else
						printf("[TX] pps: None\n");
				}
			}
			for(i = 0; i<512; i++)
			{
				if(i % 32 ==0)
					printf("\n");
				if(buf_idx[i] == 1){
					START_GRN
						printf("%d ", buf_idx[i]);
					END
				}else if(buf_idx[i] == 2){
					START_RED
						printf("%d ", buf_idx[i]);
					END
				}else if(buf_idx[i] == 3){
					START_BLU
						printf("%d ", buf_idx[i]);
					END
				}else{
					printf("%d ", buf_idx[i]);
				}
			}
			printf("\n");

			prev_pkt[0] = cur_pkt[0];
			prev_pkt[1] = cur_pkt[1];
		}
		//sleep(1); 
	}                                                                  
}

