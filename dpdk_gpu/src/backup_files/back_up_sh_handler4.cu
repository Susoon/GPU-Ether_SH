#include "hip/hip_runtime.h"
#include "sh_handler.h"

#define RING_BATCH_NUM 8
#define DPDK_RING_SIZE (BATCH_SIZE * RING_BATCH_NUM)
#define RTE_ETH_CRC_LEN 5
#define TOTAL_PKT_SIZE (PKT_SIZE + RTE_ETH_CRC_LEN)
#define ONELINE 6
#define DUMP 0

unsigned char * rx_pkt_buf;
unsigned char * tx_pkt_buf;
static int idx;
int * rx_pkt_cnt;
int tx_idx;
int * batch_size;

int leastPriority;
int greatestPriority;

static int count = 0;

__global__ void test(void)
{
	printf("Test!!!!!\n");
}

void gpu_test(void)
{
	printf("gputest!!!!!!!!!!!\n");
	hipStream_t stream;
	hipStreamCreateWithPriority(&stream, hipStreamNonBlocking, greatestPriority);
	test<<<1,1,0,stream>>>();
}

void check_error(hipError_t err)
{	
	if(err == hipSuccess)
	{
		count++;
//		printf("%dth success!!!!\n", count);
	}
	else
	{
//		printf("%s!!!!!!\n", hipGetErrorName(err));
	}
}

__global__ void print_gpu(unsigned char* d_pkt_buf)
{
	int i;
	START_RED
	printf("[GPU]:\n");
	for(i = 0; i < BATCH_SIZE; i++)
	{
		if(i != 0 && i % ONELINE == 0)
			printf("\n");
		if(i != 0 && i % PKT_SIZE == 0)
			printf("\n");
		printf("%02x ", d_pkt_buf[i]);
	}
	printf("\n");
	END
}

__device__ void mani_pkt_gpu(unsigned char * d_pkt_buf)
{
	int i;
	unsigned char tmp[6] = { 0 };

	// Swap mac
	for(i = 0; i < 6; i++){
		tmp[i] = d_pkt_buf[i];
		d_pkt_buf[i] = d_pkt_buf[i + 6];
		d_pkt_buf[i + 6] = tmp[i];
	}
	// Swap ip
	for(i = 26; i < 30; i++){
		tmp[i-26] = d_pkt_buf[i];
		d_pkt_buf[i] = d_pkt_buf[i + 4];
		d_pkt_buf[i + 4] = tmp[i-26];
	}
	// Swap port
	for(i = 34; i < 36; i++){
		tmp[i-34] = d_pkt_buf[i];
		d_pkt_buf[i] = d_pkt_buf[i + 2];
		d_pkt_buf[i + 2] = tmp[i-34];
	}	
	//Manipulatate data
	for(i = 36; i < PKT_SIZE; i++){
		d_pkt_buf[i] = 0;
	}
}

extern "C"
uint64_t monotonic_time() {
        struct timespec timespec;
        clock_gettime(CLOCK_MONOTONIC, &timespec);
        return timespec.tv_sec * ONE_SEC + timespec.tv_nsec;
}

extern "C"
void copy_to_gpu(unsigned char* buf, int size)
{
	//printf("rx_pkt_buf copy\n");
	check_error(hipMemcpy(rx_pkt_buf + (idx * BATCH_SIZE), buf, sizeof(unsigned char)* PKT_SIZE * size, hipMemcpyHostToDevice));

//	printf("size copy\n");
	check_error(hipMemcpy(batch_size, &size, sizeof(int), hipMemcpyHostToDevice));

	idx++;
	if(idx == RING_BATCH_NUM)
		idx = 0;

#if DUMP
	print_gpu<<<1,1>>>(rx_pkt_buf + (idx * BATCH_SIZE));
	hipDeviceSynchronize();
#endif
}

extern "C"
void set_gpu_mem_for_dpdk(void)
{
	size_t pkt_buffer_size = DPDK_RING_SIZE;

	idx = 0;
	tx_idx = 0;

	ASSERTRT(hipMalloc((void**)&rx_pkt_buf, pkt_buffer_size));
  	ASSERTRT(hipMemset(rx_pkt_buf, 0, pkt_buffer_size));

	ASSERTRT(hipMalloc((void**)&tx_pkt_buf, pkt_buffer_size));
  	ASSERTRT(hipMemset(tx_pkt_buf, 0, pkt_buffer_size));

	ASSERTRT(hipMalloc((void**)&rx_pkt_cnt, sizeof(int)));
  	ASSERTRT(hipMemset(rx_pkt_cnt, 0, sizeof(int)));

	ASSERTRT(hipMalloc((void**)&batch_size, sizeof(int)));
  	ASSERTRT(hipMemset(batch_size, 0, sizeof(int)));

	hipDeviceGetStreamPriorityRange(&leastPriority, &greatestPriority);

	START_GRN
	printf("[Done]____GPU mem set for dpdk____\n");
	END
}

extern "C"
int get_rx_cnt(void)
{
	int rx_cur_pkt = tx_idx;
	printf("rx_cur_pkt copy\n");
	printf("Before memcpy, rx_cur_pkt = %d\n", rx_cur_pkt);
	ASSERTRT(hipMemcpy(&rx_cur_pkt, rx_pkt_cnt, sizeof(int), hipMemcpyDeviceToHost));
	printf("After memcpy, rx_cur_pkt = %d\n", rx_cur_pkt);

	gpu_test();

//	printf("rx_pkt_cnt memset\n");
	check_error(hipMemset(rx_pkt_cnt, 0, sizeof(int)));	
	tx_idx++;

	return rx_cur_pkt;
}

extern "C"
void get_tx_buf(unsigned char* tx_buf)
{
	printf("get_tx_buf!!!!!\n");

	hipMemcpy(tx_buf, tx_pkt_buf + (tx_idx * BATCH_SIZE), sizeof(unsigned char) * BATCH_SIZE, hipMemcpyDeviceToHost);

	tx_idx++;
	if(tx_idx == RING_BATCH_NUM)
		tx_idx = 0;
}

__global__ void gpu_monitoring_loop(unsigned char * rx_pkt_buf, unsigned char * tx_pkt_buf, int * rx_pkt_cnt, int * batch_size)
{
	int mem_index = BATCH_SIZE * threadIdx.x;

	__syncthreads();
#if 1
	while(true)
	{
#if 0
		__syncthreads();
		if(rx_pkt_buf[mem_index] != 0)
		{
			__syncthreads();
			rx_pkt_buf[mem_index] = 0;

			__syncthreads();
			atomicAdd(rx_pkt_cnt, BATCH_SIZE);

			//printf("in the loop rx_pkt_cnt = %d\n", *rx_pkt_cnt);
			//mani_pkt_gpu(rx_pkt_buf + (i * PKT_SIZE));
			//memset(rx_pkt_buf + (i * PKT_SIZE), 0, PKT_SIZE); 		
					
			//memcpy(tx_pkt_buf, rx_pkt_buf, PKT_SIZE);
		}
#endif
	}
#endif
}

extern "C"
void gpu_monitor(void)
{
	hipStream_t stream;
	ASSERTRT(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
	gpu_monitoring_loop<<<1, RING_BATCH_NUM, 0, stream>>>(rx_pkt_buf, tx_pkt_buf, rx_pkt_cnt, batch_size);
}

