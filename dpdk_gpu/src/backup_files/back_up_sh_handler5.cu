#include "hip/hip_runtime.h"
#include "sh_handler.h"

#define BLOCK_NUM 64
#define THREAD_NUM BLOCK_NUM
#define RING_SIZE ((uint64_t)PKT_BATCH_SIZE * BLOCK_NUM)

#define ONELINE 6

#define DUMP 0
#define TX 0

unsigned char * rx_pkt_buf;
unsigned char * tx_pkt_buf;
static int idx;
int * rx_pkt_cnt;

int tx_idx;

int * pkt_batch_num;

extern "C"
int monotonic_time() {
        struct timespec timespec;
        clock_gettime(CLOCK_MONOTONIC, &timespec);
        return timespec.tv_sec * ONE_SEC + timespec.tv_nsec;
}

__global__ void gpu_monitor(unsigned char * rx_pkt_buf, int * rx_pkt_cnt, int * pkt_batch_num);

#if DUMP

__global__ void print_gpu(unsigned char* d_pkt_buf, int * pkt_num)
{
	int i;
	int total_pkt_num = *pkt_num * PKT_SIZE;
	START_RED
	printf("[GPU]: pkt_num = %d\n", *pkt_num);
	for(i = 0; i < total_pkt_num; i++)
	{
		if(i != 0 && i % ONELINE == 0)
			printf("\n");
		if(i != 0 && i % PKT_SIZE == 0)
			printf("\n");
		printf("%02x ", d_pkt_buf[i]);
	}
	printf("\n\n");
	END
}

#endif

__device__ void mani_pkt_gpu(unsigned char * d_pkt_buf)
{
	int i;
	unsigned char tmp[6] = { 0 };

	// Swap mac
	for(i = 0; i < 6; i++){
		tmp[i] = d_pkt_buf[i];
		d_pkt_buf[i] = d_pkt_buf[i + 6];
		d_pkt_buf[i + 6] = tmp[i];
	}
	// Swap ip
	for(i = 26; i < 30; i++){
		tmp[i-26] = d_pkt_buf[i];
		d_pkt_buf[i] = d_pkt_buf[i + 4];
		d_pkt_buf[i + 4] = tmp[i-26];
	}
	// Swap port
	for(i = 34; i < 36; i++){
		tmp[i-34] = d_pkt_buf[i];
		d_pkt_buf[i] = d_pkt_buf[i + 2];
		d_pkt_buf[i + 2] = tmp[i-34];
	}	
	//Manipulatate data
	for(i = 36; i < PKT_SIZE; i++){
		d_pkt_buf[i] = 0;
	}
}

extern "C"
int copy_to_gpu(unsigned char* buf, int pkt_num)
{

	ASSERTRT(hipMemcpy(rx_pkt_buf + (idx * PKT_BATCH_SIZE), buf, sizeof(unsigned char) * pkt_num * PKT_SIZE, hipMemcpyHostToDevice));

	hipMemcpy(pkt_batch_num + idx, &pkt_num, sizeof(int), hipMemcpyHostToDevice);
#if 0
	hipStream_t stream;
	ASSERTRT(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
	gpu_monitor<<<1, THREAD_NUM, 0, stream>>>(rx_pkt_buf, rx_pkt_cnt, pkt_batch_num);
	hipDeviceSynchronize();
	hipStreamDestroy(stream);
#endif

#if DUMP
	print_gpu<<<1,1>>>(rx_pkt_buf + (idx * PKT_BATCH_SIZE), pkt_batch_num + idx);
	hipDeviceSynchronize();
#endif

	idx++;
	if(idx == BLOCK_NUM)
		idx = 0;
	
	return 1;
}

extern "C"
void set_gpu_mem_for_dpdk(void)
{
	idx = 0;
	tx_idx = 0;

	START_BLU
	printf("RING_SIZE = %d\n", RING_SIZE);
	printf("PKT_SIZE = %d, PKT_BATCH = %d + %d\n", PKT_SIZE, PKT_BATCH - RX_NB, RX_NB);
	END

	ASSERTRT(hipMalloc((void**)&rx_pkt_buf, RING_SIZE));
  	ASSERTRT(hipMemset(rx_pkt_buf, 0, RING_SIZE));

//	ASSERTRT(hipMalloc((void**)&tx_pkt_buf, RING_SIZE));
// 	ASSERTRT(hipMemset(tx_pkt_buf, 0, RING_SIZE));

	ASSERTRT(hipMalloc((void**)&rx_pkt_cnt, sizeof(int)));
  	ASSERTRT(hipMemset(rx_pkt_cnt, 0, sizeof(int)));

	ASSERTRT(hipMalloc((void**)&pkt_batch_num, sizeof(int) * BLOCK_NUM));
  	ASSERTRT(hipMemset(pkt_batch_num, 0, sizeof(int) * BLOCK_NUM));

	START_GRN
	printf("[Done]____GPU mem set for dpdk____\n");
	END
}

extern "C"
int get_rx_cnt(void)
{
	int rx_cur_pkt = 0;

	hipMemcpy(&rx_cur_pkt, rx_pkt_cnt, sizeof(int), hipMemcpyDeviceToHost);

	hipMemset(rx_pkt_cnt, 0, sizeof(int));	
	printf("rx_cur_pkt = %d\n", rx_cur_pkt);

	return rx_cur_pkt;
}

extern "C"
int get_tx_buf(unsigned char* tx_buf)
{
	int tx_cur_pkt = 0;

	hipMemcpy(tx_buf, tx_pkt_buf + (tx_idx * PKT_BATCH_SIZE), sizeof(unsigned char) * PKT_BATCH_SIZE, hipMemcpyDeviceToHost);

//	hipMemcpy(&tx_cur_pkt, pkt_batch_num + tx_idx, sizeof(int), hipMemcpyDeviceToHost);

	tx_idx++;
	if(tx_idx == BLOCK_NUM)
		tx_idx = 0;

	return tx_cur_pkt;
}

__global__ void gpu_monitor(unsigned char * rx_pkt_buf, int * rx_pkt_cnt, int * pkt_batch_num)
{
	int mem_index = PKT_BATCH_SIZE * threadIdx.x;

	__syncthreads();
	if(pkt_batch_num[threadIdx.x] != 0 && rx_pkt_buf[mem_index + ((pkt_batch_num[threadIdx.x] - 1) * PKT_SIZE)] != 0)
	{
		__syncthreads();
		rx_pkt_buf[mem_index + ((pkt_batch_num[threadIdx.x] - 1) * PKT_SIZE)] = 0;

		__syncthreads();
		atomicAdd(rx_pkt_cnt, pkt_batch_num[threadIdx.x]);
		
#if TX
		__syncthreads();
		memcpy(tx_pkt_buf + mem_index, rx_pkt_buf + mem_index, PKT_BATCH_SIZE);
/*
		for(int i = 0; i < pkt_batch_num[threadIdx.x]; i++)
		{
			__syncthreads();
			mani_pkt_gpu(tx_pkt_buf + mem_index + i * PKT_SIZE);
		}
*/
#endif
		__syncthreads();
		memset(pkt_batch_num + threadIdx.x, 0, sizeof(int));
		
	}
}

#if 0
__global__ void gpu_mani_loop(unsigned char * tx_pkt_buf,int * pkt_batch_num)
{
	__syncthreads();
	if(pkt_batch_num[threadIdx.x] != 0 && rx_pkt_buf[mem_index + ((pkt_batch_num[threadIdx.x] - 1) * PKT_SIZE)] != 0)
	{
		__syncthreads();
		memcpy(tx_pkt_buf + mem_index, rx_pkt_buf + mem_index, PKT_BATCH_SIZE);

		__syncthreads();
		mani_pkt_gpu(tx_pkt_buf + mem_index + i * PKT_SIZE);
	}
}	
#endif

extern "C"
void gpu_monitor_loop(void)
{
	hipStream_t stream;
	ASSERTRT(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
	while(true)
	{
		gpu_monitor<<<1, THREAD_NUM, 0, stream>>>(rx_pkt_buf, rx_pkt_cnt, pkt_batch_num);
		hipDeviceSynchronize();
	}
}

