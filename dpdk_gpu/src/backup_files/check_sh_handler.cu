#include "hip/hip_runtime.h"
#include "sh_handler.h"

#define DPDK_RING_SIZE 2 * 1024 * 1024 //2MB
#define PKT_SIZE 64
#define RTE_ETH_CRC_LEN 4
#define TOTAL_PKT_SIZE (PKT_SIZE + RTE_ETH_CRC_LEN)

#define ONELINE 6

#define RING_CHECK 0

unsigned char * pinned_pkt_buf;
unsigned char * tmp;
unsigned char * cpy_buf;
static int idx;

__global__ void Check_gpu(void)
{
	printf("\n\n\n\n\n\n");
	printf("____________GPU function is called______________\n");
	printf("\n\n\n\n\n\n");
}

void Check_buf(unsigned char * buf)
{
	printf("\n%dth pkt_dump: \n", idx);
	for(int i = 0; i < DPDK_RING_SIZE; i +=  0x1000)
	{
		if(i % (0x1000 * 32) == 0)
			printf("\n");
		if(i / 0x1000 == idx)
			START_GRN
		printf("%02x ", buf[i]);
		END
	}
	printf("\n");
}

void Dump_fct(unsigned char * buf, int size)
{
	printf("%dth pkt_dump: \n", idx);
	for(int i = 0; i < TOTAL_PKT_SIZE; i++){
		if(i != 0 && i % ONELINE == 0)
			printf("\n");
		printf("%02x ", buf[i]);
		}
	printf("\n");
}

__global__ void print_gpu(unsigned char* d_pkt_buf)
{
	int i;
	START_RED
	printf("[GPU]:\n");
	for(i = 0; i < TOTAL_PKT_SIZE; i++)
	{
		if(i != 0 && i % ONELINE == 0)
			printf("\n");
		printf("%02x ", d_pkt_buf[i]);
	}
	printf("\n");
	END
}

#if RING_CHECK

extern "C" 
void copy_to_gpu(unsigned char* buf, int size)
{
	memcpy(pinned_pkt_buf + (idx * 0x1000), buf, size);

	//Dump_fct(pinned_pkt_buf, size);
	print_gpu<<<1,1>>>(pinned_pkt_buf + (idx * 0x1000));
	
	Check_buf(pinned_pkt_buf);	
	memcpy(pinned_pkt_buf + (idx * 0x1000), zero_arr, size);

	idx++;
	if(idx == 512)
		idx = 0;
}


extern "C"
void set_gpu_mem_for_dpdk(void)
{
	size_t pkt_buffer_size = DPDK_RING_SIZE;
	idx = 0;

	pinned_pkt_buf = (unsigned char*)calloc(pkt_buffer_size, sizeof(unsigned char));
	zero_arr = (unsigned char*)calloc(pkt_buffer_size, sizeof(unsigned char));

	START_GRN
	printf("[Done]____GPU mem set for dpdk____\n");
	END
}

#else

extern "C"
void copy_to_gpu(unsigned char* buf, int size)
{
	hipMemcpy(pinned_pkt_buf + (idx * 0x1000), buf, sizeof(unsigned char)*size, hipMemcpyHostToDevice);
	
	print_gpu<<<1,1>>>(pinned_pkt_buf + (idx * 0x1000));
	hipDeviceSynchronize();

	idx++;
	if(idx == 512)
		idx = 0;
	hipMemcpy(buf, pinned_pkt_buf + (idx * 0x1000), sizeof(unsigned char) * size, hipMemcpyDeviceToHost);
}

extern "C"
void set_gpu_mem_for_dpdk(void)
{
	size_t pkt_buffer_size = DPDK_RING_SIZE;
	idx = 0;
	ASSERTRT(hipMalloc((void**)&pinned_pkt_buf, pkt_buffer_size));
  	ASSERTRT(hipMemset(pinned_pkt_buf, 0, pkt_buffer_size));

	ASSERTRT(hipMalloc((void**)&tmp, sizeof(unsigned char) * 6));
  	ASSERTRT(hipMemset(tmp, 0, 6));

	cpy_buf = (unsigned char*)calloc(pkt_buffer_size, sizeof(unsigned char));

	START_GRN
	printf("[Done]____GPU mem set for dpdk____\n");
	END
}

#endif

extern "C"
void cudasynch(void)
{
	hipDeviceSynchronize();
}

