#include "hip/hip_runtime.h"
#include "sh_handler.h"

#define DPDK_RING_SIZE (BATCH_SIZE * 8)
#define RTE_ETH_CRC_LEN 5
#define TOTAL_PKT_SIZE (PKT_SIZE + RTE_ETH_CRC_LEN)
#define ONELINE 6

#define MANI 1

unsigned char * pinned_pkt_buf;
unsigned char * tmp;
static int idx;

extern "C"
uint64_t monotonic_time() {
        struct timespec timespec;
        clock_gettime(CLOCK_MONOTONIC, &timespec);
        return timespec.tv_sec * ONE_SEC + timespec.tv_nsec;
}

__global__ void mani_pkt_gpu(unsigned char * d_pkt_buf, unsigned char * tmp, int size)
{
	int i;
	// Swap mac
	for(i = 0; i < 6; i++){
		tmp[i] = d_pkt_buf[i];
		d_pkt_buf[i] = d_pkt_buf[i + 6];
		d_pkt_buf[i + 6] = tmp[i];
	}
	// Swap ip
	for(i = 26; i < 30; i++){
		tmp[i-26] = d_pkt_buf[i];
		d_pkt_buf[i] = d_pkt_buf[i + 4];
		d_pkt_buf[i + 4] = tmp[i-26];
	}
	// Swap port
	for(i = 34; i < 36; i++){
		tmp[i-34] = d_pkt_buf[i];
		d_pkt_buf[i] = d_pkt_buf[i + 2];
		d_pkt_buf[i + 2] = tmp[i-34];
	}
	// Manipulatate data
	for(i = 36; i < size; i++){
		d_pkt_buf[i] = 0;
	}
}	

__global__ void print_gpu(unsigned char* d_pkt_buf)
{
	int i;
	START_RED
	printf("[GPU]:\n");
	for(i = 0; i < TOTAL_PKT_SIZE; i++)
	{
		if(i != 0 && i % ONELINE == 0)
			printf("\n");
		printf("%02x ", d_pkt_buf[i]);
	}
	printf("\n");
	END
}

extern "C"
void copy_to_gpu(unsigned char* buf)
{
	hipMemcpy(pinned_pkt_buf + (idx * BATCH_SIZE), buf, sizeof(unsigned char)*BATCH_SIZE, hipMemcpyHostToDevice);

#if MANI	

	mani_pkt_gpu<<<1,1>>>(pinned_pkt_buf + (idx * BATCH_SIZE), tmp, BATCH_SIZE);
	print_gpu<<<1,1>>>(pinned_pkt_buf + (idx * BATCH_SIZE));
	hipDeviceSynchronize();
	hipMemcpy(buf, pinned_pkt_buf + (idx * BATCH_SIZE), sizeof(unsigned char) * BATCH_SIZE, hipMemcpyDeviceToHost);

#endif

	idx++;
	if(idx == 512)
		idx = 0;
}

extern "C"
void set_gpu_mem_for_dpdk(void)
{
	size_t pkt_buffer_size = DPDK_RING_SIZE;
	idx = 0;
	ASSERTRT(hipMalloc((void**)&pinned_pkt_buf, pkt_buffer_size));
  	ASSERTRT(hipMemset(pinned_pkt_buf, 0, pkt_buffer_size));

	ASSERTRT(hipMalloc((void**)&tmp, sizeof(unsigned char) * 6));
  	ASSERTRT(hipMemset(tmp, 0, 6));

	START_GRN
	printf("[Done]____GPU mem set for dpdk____\n");
	END
}

extern "C"
void cudasynch(void)
{
	hipDeviceSynchronize();
}

