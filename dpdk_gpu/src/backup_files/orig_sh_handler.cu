#include "hip/hip_runtime.h"
#include "sh_handler.h"

#define DPDK_RING_SIZE 2 * 1024 * 1024 //2MB
#define PKT_SIZE 64
#define RTE_ETH_CRC_LEN 4
#define TOTAL_PKT_SIZE (PKT_SIZE + RTE_ETH_CRC_LEN)

#define ONELINE 6

unsigned char * pinned_pkt_buf;
static int idx;

/*
__device__ uint8_t tmp_pkt[60] = {\
0x00, 0x1b, 0x21, 0xbc, 0x11, 0x52, 0xa0, 0x36, 0x9f, 0x03, 0x13, 0x86, 0x08, 0x00, 0x45, 0x10,\
0x00, 0x2e, 0x00, 0x00, 0x40, 0x00, 0x40, 0x11, 0x00, 0x00, 0x0a, 0x00, 0x00, 0x03, 0x0a, 0x00,\
0x00, 0x04, 0x04, 0xd2, 0x04, 0xd2, 0x00, 0x1a, 0x2c, 0xd6, 0x6f, 0x98, 0x26, 0x35, 0x02, 0xc9,\
0x83, 0xd7, 0x8b, 0xc3, 0xf7, 0xb5, 0x20, 0x8d, 0x48, 0x8d, 0xc0, 0x36};
*/

/* Suhwan pinning buffer 02/06 */

void Dump_fct(unsigned char * buf, int size)
{
	printf("%dth pkt_dump: \n", idx);
	for(int i = 0; i < TOTAL_PKT_SIZE; i++){
		if(i != 0 && i % ONELINE == 0)
			printf("\n");
		printf("%02x ", buf[i]);
		}
	printf("\n");
}


int sh_pin_buffer(void)
{
	int ret = 0;
	int retcode;

	retcode = hipHostAlloc((void**)&pinned_pkt_buf, sizeof(unsigned char) * TOTAL_PKT_SIZE, hipHostMallocDefault);
	if(retcode == hipErrorOutOfMemory)
	{
		ret = errno;
		printf("hipHostAlloc error (errno=%d)\n", ret);
	}

    return ret;
}

__global__ void print_gpu(unsigned char* d_pkt_buf)
{
	int i;
	printf("[GPU]:\n");
	for(i = 0; i < TOTAL_PKT_SIZE; i++)
	{
		if(i != 0 && i % ONELINE == 0)
			printf("\n");
		printf("%02x ", d_pkt_buf[i]);
	}
	printf("\n");
}

void copy_to_pinned_buffer(unsigned char * d_pkt_buf, int size)
{
	printf("___1___________copy_to_pinned_buffer___\n");
	hipMemcpy(pinned_pkt_buf, d_pkt_buf, size, hipMemcpyDeviceToDevice);		
	printf("___2___________copy_to_pinned_buffer___\n");
}

#if 0
extern "C" 
void copy_to_gpu(unsigned char* buf, int size)
{
	unsigned char * d_pkt_buf;
	hipMalloc((void**)&d_pkt_buf, sizeof(unsigned char) * size);
	//printf("____1__________copy_to_gpu____\n");
	hipMemcpy(d_pkt_buf, buf, sizeof(unsigned char)*size, hipMemcpyHostToDevice);
	print_gpu<<<1,1>>>(d_pkt_buf);
	//printf("____2__________copy_to_gpu____\n");
	hipFree(d_pkt_buf);
}
#endif

extern "C" 
void copy_to_gpu(unsigned char* buf, int size)
{
	//printf("____1__________copy_to_gpu____\n");
	hipMemcpy(pinned_pkt_buf + (idx * 0x1000), buf, sizeof(unsigned char)*size, hipMemcpyHostToDevice);
	//Dump_fct(buf, size);	
	print_gpu<<<1,1>>>(pinned_pkt_buf + (idx * 0x1000));
	idx++;
	if(idx == 512)
		idx = 0;
	//printf("____2__________copy_to_gpu____\n");
}

extern "C"
void set_gpu_mem_for_dpdk(void)
{
	size_t pkt_buffer_size = DPDK_RING_SIZE;
	idx = 0;
	ASSERTRT(hipMalloc((void**)&pinned_pkt_buf, pkt_buffer_size));
  	ASSERTRT(hipMemset(pinned_pkt_buf, 0, pkt_buffer_size));

	//pinned_pkt_buf = d_pkt_buf;
	//printf("pinned_pkt_buf = %p\n", pinned_pkt_buf);
	START_GRN
	printf("[Done]____GPU mem set for dpdk____\n");
	END
}

__device__ void print_pinned_buffer(unsigned char* d_pkt_buf)
{
	int i;
	printf("[Pinned Buffer]:\n");
	for(i = 0; i < TOTAL_PKT_SIZE; i++)
	{
		if(i != 0 && i % ONELINE == 0)
			printf("\n");
		printf("%02x ", d_pkt_buf[i]);
	}
	printf("\n");
}

__global__ void read_loop(unsigned char* d_pkt_buf)
{
	while(1)
	{
		print_pinned_buffer(d_pkt_buf);
	}

}

extern "C"
void read_handler(void)
{

  //set_gpu_mem_for_dpdk();
  sh_pin_buffer();
  read_loop<<<1,1>>>(pinned_pkt_buf);
}
