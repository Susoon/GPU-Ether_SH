#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#define START_RED printf("\033[1;31m");
#define START_GRN printf("\033[1;32m");
#define START_YLW printf("\033[1;33m");
#define START_BLU printf("\033[1;34m");
#define END printf("\033[0m");

#define ONE_SEC (1000 * 1000 * 1000)
#define HALF (1024 * 32 * 64)

#define NORM 0

#define LOOP 0

#define SEL 1

#define CASE 17

#define REVERSE 0

FILE * data = fopen("data.txt", "r");

uint64_t latency[17] = { 0 };
const char* size_str[17] = { "64", "128", "256", "512", "1024", "1514",\
		 "64 * 32", "64 * 64", "64 * 128", "64 * 256",	\
		"64 * 512", "64 * 1024", "64 * 1024 * 2", "64 * 1024 * 4",		 	\
		"64 * 1024 * 8", "64 * 1024 * 16", "64 * 1024 * 32"};
int size[17] = { 64, 128, 256, 512, 1024, 1514, 64 * 32, 64 * 64,\
			 64 * 128, 64 * 256, 64 * 512, 64 * 1024,\
			 64 * 1024 * 2, 64 * 1024 * 4, 64 * 1024 * 8,\
			 64 * 1024 * 16, 64 * 1024 * 32};

int start[17] = { 0 };
int end[17] = { 0 };

int monotonic_time() 
{
	struct timespec timespec;
	clock_gettime(CLOCK_MONOTONIC, &timespec);
	return timespec.tv_sec * ONE_SEC + timespec.tv_nsec;
}

void call_data(char * host_buf, int size)
{
	fseek(data, 0, SEEK_SET);
	fgets(host_buf, sizeof(char) * size, data);
/*
	for(int i = 0; i < size; i++)
	{
		fscanf(data, "%c", host_buf + i);
	}
*/
}

void once(char * device_buf, char * host_buf, int test_cnt)
{
	int i = 0;
	
	int skip = 0;

	while(i < test_cnt)
	{
		call_data(host_buf, HALF * 2);
		skip = 0;
#if REVERSE
		for(int j = CASE - 1; j >= 0; j--)
#else
		for(int j = 0; j < CASE; j++)
#endif
		{
#if NORM
			skip += size[j];
#else
			skip = rand();
#endif
			if(skip >= HALF)
				skip %= HALF;
			start[j] = monotonic_time();
			hipMemcpy(device_buf, host_buf + skip, size[j], hipMemcpyHostToDevice);
			end[j] = monotonic_time();
			latency[j] += end[j] - start[j];
			hipMemset(device_buf, 0, size[j]);
		}
		i++;
	}

	for(i = 0; i < CASE; i++)
	{
		latency[i] /= (uint64_t)test_cnt;
	}
}

void loop(char * device_buf, char * host_buf, int test_cnt, int loop_cnt)
{
	int i = 0;

	int skip = 0;

	while(i < test_cnt)
	{
		call_data(host_buf, HALF * 2);
		skip = 0;
#if REVERSE
		for(int j = CASE - 1; j >= 0; j--)
#else
		for(int j = 0; j < CASE; j++)
#endif
		{
#if NORM
			skip += size[j];
#else
			skip = rand();
#endif
			if(skip >= HALF)
				skip %= HALF;
			start[j] = monotonic_time();
			for(int k = 0; k < loop_cnt; k++)
			{
			hipMemcpy(device_buf, host_buf + skip, size[j], hipMemcpyHostToDevice);
			}
			end[j] = monotonic_time();
			latency[j] += (end[j] - start[j]) / (uint64_t)loop_cnt;
		}
		i++;
	}

	for(i = 0; i < CASE; i++)
	{
		latency[i] /= (uint64_t)test_cnt;
	}
}

void same_cnt_loop(char * device_buf, char * host_buf, int test_cnt)
{
	int i = 0;

	int loop_cnt = size[16];
	int cur_loop_cnt;
	int skip = 0;

	while(i < test_cnt)
	{
		call_data(host_buf, HALF * 2);
		skip = 0;
#if REVERSE
		for(int j = CASE - 1; j >= 0; j--)
#else
		for(int j = 0; j < CASE; j++)
#endif
		{
			cur_loop_cnt = loop_cnt / size[j];
#if NORM
			skip += size[j];
#else
			skip = rand();
#endif
			if(skip >= HALF)
				skip %= HALF;
			start[j] = monotonic_time();
			for(int k = 0; k < cur_loop_cnt; k++)
			{
			hipMemcpy(device_buf, host_buf + skip, size[j], hipMemcpyHostToDevice);
			}
			end[j] = monotonic_time();
			latency[j] += end[j] - start[j];
		}
		i++;
	}

	for(i = 0; i < CASE; i++)
	{
		latency[i] /= (uint64_t)test_cnt;
	}
}

void print_result(int test_cnt)
{
	START_RED
	printf("\n\n______________________TEST START_______________________\n\n");
	END

	START_YLW
#if NORM
	printf("   NORMAL DATA TEST!\n");
#else
	printf("   RANDOM DATA TEST!\n");
#endif
	END

	START_GRN
#if LOOP
	printf("   TEST WAS RUNNED %d TIMES!\n", test_cnt);
#elif SEL
	printf("   TEST WAS RUNNED SAME TIMES!\n");
#else
	printf("   TEST WAS RUNNED ONCE!\n");
#endif
	END

#if REVERSE
	START_BLU
	printf("   TEST WAS RUNNED REVERSED ORDER!\n");
	END
#endif

	for(int i = 0; i < CASE; i++)
	{
		printf("   data size : %s, latency : %ld\n", size_str[i], latency[i]);
	}

	START_RED
	printf("\n______________________TEST END_______________________\n\n\n");
	END
}


int main(void)
{
	int test_cnt;

	char * device_buf;
	char * host_buf;

	srand(time(NULL));

	host_buf = (char *)calloc(HALF * 2, sizeof(char));
	hipHostAlloc((void**)&device_buf, HALF * sizeof(char), hipHostMallocDefault);
	hipMemset(device_buf, 0 ,HALF * sizeof(char)); 

	printf("Enter the test_cnt\n");
	scanf("%d", &test_cnt);

#if LOOP
	int loop_cnt;

	printf("Enter the loop_cnt\n");
	scanf("%d", &loop_cnt);
	loop(device_buf, host_buf, test_cnt, loop_cnt);
#elif SEL
	same_cnt_loop(device_buf, host_buf, test_cnt);
#else
	once(device_buf, host_buf, test_cnt);
#endif
	print_result(test_cnt);

	hipFree(device_buf);

	fclose(data);

	return 0;
}
