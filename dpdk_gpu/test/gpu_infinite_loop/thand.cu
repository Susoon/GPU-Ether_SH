//#include "thand.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <pthread.h>

#define CYCLE 1024 * 1024 * 1024
#define THREAD 1

int * count;

__global__ void Check_gpu(int * count)
{
#if 1
		__syncthreads();
	while(*count < CYCLE)
	{
		//printf("\n\n\n\n\n\n");
		//printf("____________GPU function is called______________\n");
		__syncthreads();
		atomicAdd(count, 1);
		//printf("\n\n\n\n\n\n");
	}
	printf("_______________Out of GPU fct loop________________\n");
#else
		printf("____________GPU function is called______________\n");
#endif
}

extern "C"
void Check(void)
{
	printf("Check!!\n");
	Check_gpu<<<1,512>>>(count);
	hipDeviceSynchronize();
	printf("End Check!!!!!!!!!\n");
}

extern "C"
void cudasynch(void)
{
	hipDeviceSynchronize();
}

#if THREAD
void* get_cnt(void * data)
{
	int prev = 0;
	printf("In get_cnt!!!!!!!!!!!!!!!\n");
	while(1)
	{
		int ret = 0;
		hipMemcpy(&ret, count, sizeof(int), hipMemcpyDeviceToHost);
		if(prev != ret)
		{
			printf("In CPU : count = %d\n", ret);
			prev = ret;
		}
	}
}

#else

void get_cnt(void)
{
	int j = 0;
	printf("In get_cnt!!!!!!!!!!!!!!!\n");
	while(j < 100)
	{
		int ret = 0, tmp;
		cudaMemcpy(&ret, count, sizeof(int), cudaMemcpyDeviceToHost);
		printf("In CPU : count = %d\n", ret);
		j++;
	}
}

#endif

int main(void)
{
	pthread_t thread;

	hipMalloc((void**)&count, sizeof(int));
	hipMemset(count, 0, sizeof(int));

	printf("___1____\n");
#if THREAD
	pthread_create(&thread, NULL, get_cnt, NULL);
	Check();
	printf("___2____\n");
#else
	Check();
	get_cnt();
	printf("___2____\n");
#endif
	hipDeviceSynchronize();
	printf("___3____\n");
	return 0;
}
